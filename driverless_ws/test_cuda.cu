#include <cstdio>
#include <hip/hip_runtime.h>  // Include the CUDA runtime header

int main() {
    int n; 
    hipError_t err = hipGetDeviceCount(&n);  // Capture return status

    if (err != hipSuccess) {
        printf("hipGetDeviceCount failed: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Found %d CUDA device%s\n", n, (n == 1 ? "" : "s"));
    return 0;
}
