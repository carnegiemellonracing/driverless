#include "hip/hip_runtime.h"
#include <cassert>
#include <glad/glad.h>
#include <cuda_gl_interop.h>

#include <cuda_utils.cuh>

constexpr dim3 block_dim {32, 32, 1};

__global__ void view_tex(hipTextureObject_t texture) {
    if (threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
        const float4 col = tex2D<float4>(texture, 0.5, 0.5);
        printf("Color: %f %f %f %f", col.x, col.y, col.z, col.w);
    }
}

void cuda_test(GLuint rbo, uint width, uint height) {
    hipGraphicsResource_t img_rsc;
    CUDA_CALL(hipGraphicsGLRegisterImage(&img_rsc, rbo, GL_RENDERBUFFER, hipGraphicsRegisterFlagsNone));
    CUDA_CALL(hipGraphicsMapResources(1, &img_rsc));

    hipResourceDesc img_rsc_desc {};
    img_rsc_desc.resType = hipResourceTypeMipmappedArray;
    CUDA_CALL(cudaGraphicsResourceGetMappedMipmappedArray(&img_rsc_desc.res.mipmap.mipmap, img_rsc));

    hipTextureDesc img_tex_desc {};
    img_tex_desc.addressMode[0] = hipAddressModeClamp;
    img_tex_desc.addressMode[1] = hipAddressModeClamp;
    img_tex_desc.filterMode = hipFilterModeLinear;
    img_tex_desc.readMode = hipReadModeElementType;
    img_tex_desc.normalizedCoords = true;

    hipTextureObject_t tex;
    hipCreateTextureObject(&tex, &img_rsc_desc, &img_tex_desc, nullptr);

    view_tex<<<dim3(width / block_dim.x, height / block_dim.y, 1), block_dim>>>(tex);

    CUDA_CALL(hipGraphicsUnmapResources(1, &img_rsc));
}