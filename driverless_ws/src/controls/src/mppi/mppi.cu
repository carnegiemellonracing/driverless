#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/sequence.h>
#include <thrust/random.h>

#include <cuda_globals/cuda_globals.cuh>
#include <cmath>

#include "cuda_utils.cuh"
#include "mppi.cuh"
#include "functors.cuh"


namespace controls {
    namespace mppi {

        std::unique_ptr<MppiController> MppiController::create() {
            return std::make_unique<MppiController_Impl>();
        }

        MppiController_Impl::MppiController_Impl() {
            m_action_trajectories = thrust::device_malloc<float>(num_action_trajectories);
            m_cost_to_gos = thrust::device_malloc<float>(num_timesteps * num_samples);
            m_last_action_trajectory = thrust::device_malloc<float>(num_timesteps * action_dims);
        }

        MppiController_Impl::~MppiController_Impl() {
            thrust::device_free(m_action_trajectories);
            thrust::device_free(m_cost_to_gos);
            thrust::device_free(m_last_action_trajectory);
        }

        Action MppiController_Impl::generate_action() {
            // swap device states
            cuda_globals::lock_and_swap_state_buffers();

            // call kernels
            generate_brownians();
            populate_cost();

            // not actually on device, just still in a device action struct
            DeviceAction dev_action = reduce_actions();
//            // copy action to host, return
//            return controls::Action();
            Action action;
            std::copy(std::begin(dev_action.data), std::end(dev_action.data), action.begin());
            return action;
        }

        // Private member functions of the controller
        void prefix_scan(thrust::device_ptr<float> normal) {
            auto actions = thrust::device_pointer_cast((DeviceAction*)normal.get());
            auto keys = thrust::make_transform_iterator(thrust::make_counting_iterator(0), DivBy<num_timesteps> {});

            thrust::inclusive_scan_by_key(keys, keys + num_samples * num_timesteps,
                                          actions, actions,
                                          Equal<size_t> {}, AddActions {});
        }

        void MppiController_Impl::generate_brownians() {

            // create the random generator
            hiprandGenerator_t rng;
            CURAND_CALL(hiprandCreateGenerator(&rng, rng_type));
            CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(rng, seed));

            // generate normals, put it in device memory pointed to by m_action_trajectories
            CURAND_CALL(hiprandGenerateNormal(rng, m_action_trajectories.get(), num_action_trajectories, 0, 1));

            // make the normals brownian
            thrust::counting_iterator<size_t> indices {0};
            thrust::for_each(indices, indices + num_action_trajectories, TransformStdNormal {m_action_trajectories});
            prefix_scan(m_action_trajectories);

            // clean up memory
            CURAND_CALL(hiprandDestroyGenerator(rng));
        }


        DeviceAction MppiController_Impl::reduce_actions() {
            thrust::device_vector<DeviceAction> averaged_actions (num_timesteps);
            thrust::counting_iterator<uint32_t> indices {0};

            thrust::for_each(indices, indices + num_timesteps, ReduceTimestep {
                averaged_actions.data().get(),
                m_action_trajectories.get(),
                m_cost_to_gos.get()
            });

            thrust::host_vector<DeviceAction> averaged_actions_host = averaged_actions;
            DeviceAction res;

            // copy averaged action into result for returning
            for (int i = 0; i < action_dims; i++) {
                res.data[i] = averaged_actions_host.data()[0].data[i];
            }
            return res;
        }

        void MppiController_Impl::populate_cost() {
            thrust::counting_iterator<uint32_t> indices{0};
            PopulateCost populate_cost {m_action_trajectories, m_action_trajectories,
                                        m_cost_to_gos, m_last_action_trajectory, thrust::device_pointer_cast(cuda_globals::curr_state_read)};
            thrust::for_each(indices, indices + num_samples, populate_cost);
        }
    }
}