#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <mutex>
#include <cuda_globals/cuda_globals.cuh>
#include <cmath>
#include <utils/cuda_utils.cuh>

#include "mppi.cuh"
#include "functors.cuh"


namespace controls {
    namespace mppi {

        std::shared_ptr<MppiController> MppiController::create(std::mutex& mutex, LoggerFunc logger) {
            return std::make_shared<MppiController_Impl>(mutex, logger);
        }

        MppiController_Impl::MppiController_Impl(std::mutex& mutex, LoggerFunc logger)
            : m_action_trajectories(num_action_trajectories),
              m_cost_to_gos(num_samples * num_timesteps),
              m_log_prob_densities(num_samples * num_timesteps),
              m_action_weight_tuples(num_samples * num_timesteps),
              m_rng(),
              m_last_action {},
#ifdef DISPLAY
              m_state_trajectories(num_samples * num_timesteps * state_dims),
              m_last_curr_state {},
#endif
              m_last_action_trajectory(num_timesteps - 1),
              m_logger {logger},
              m_mutex (mutex) {  // -1 because last element will always be
                                                                             // inferred from second to last
            for (uint32_t i = 0; i < num_timesteps - 1; i++) {
                DeviceAction to_send;
                for (int j = 0; j < action_dims; j++) {
                    to_send.data[j] = init_action_trajectory[i * action_dims + j];
                }
                m_last_action_trajectory[i] = to_send;
            }

            CURAND_CALL(hiprandCreateGenerator(&m_rng, rng_type));
            CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(m_rng, seed));
        }

        MppiController_Impl::~MppiController_Impl() {
            CURAND_CALL(hiprandDestroyGenerator(m_rng));
        }

        Action MppiController_Impl::generate_action() {
            std::lock_guard<std::mutex> guard {m_mutex};

            // call kernels
            m_logger("generating brownians");
            generate_brownians();

            m_logger("generating Log Probability Densities");
            generate_log_probability_density();

            m_logger("populating cost");
            populate_cost();

            m_logger("generating action weight tuples");
            generate_action_weight_tuples();

            m_logger("reducing actions");
            // not actually on device, just still in a device action struct
            thrust::device_vector<DeviceAction> averaged_trajectory = reduce_actions();

            DeviceAction host_action = m_last_action * action_momentum + (1 - action_momentum) * averaged_trajectory[0];

            Action result_action;
            std::copy(
                std::begin(host_action.data), std::end(host_action.data),
                result_action.begin()
            );

            thrust::copy(
                averaged_trajectory.begin() + 1,
                averaged_trajectory.end(),
                m_last_action_trajectory.begin()
            );

            m_last_action = host_action;

#ifdef DISPLAY
            CUDA_CALL(hipMemcpyFromSymbol(
                m_last_curr_state.data(),
                cuda_globals::curr_state,
                state_dims * sizeof(float)
            ));
#endif

            return result_action;
        }

        void MppiController_Impl::set_logger(LoggerFunc logger) {
            std::lock_guard<std::mutex> guard {m_mutex};

            m_logger = logger;
        }

#ifdef DISPLAY
        std::vector<float> MppiController_Impl::last_state_trajectories(uint32_t num) {
            std::lock_guard<std::mutex> guard {m_mutex};
            const uint32_t num_floats = num * state_dims * num_timesteps;

            std::vector<float> result (num_floats);
            thrust::copy(m_state_trajectories.begin(), m_state_trajectories.begin() + num_floats, result.begin());

            return result;
        }

        std::vector<glm::fvec2> MppiController_Impl::last_reduced_state_trajectory() {
            std::lock_guard<std::mutex> guard {m_mutex};

            std::vector<glm::fvec2> result (m_last_action_trajectory.size() + 1);

            DeviceAction action = m_last_action;

            State state = m_last_curr_state;
            result[0] = {state[state_x_idx], state[state_y_idx]};

            size_t i = 0;
            while (i < m_last_action_trajectory.size() + 1) {
                if (i >= 1) {
                    action = m_last_action_trajectory[i - 1];
                }

                ONLINE_DYNAMICS_FUNC(state.data(), action.data, state.data(), controller_period);

                result[i] = {state[state_x_idx], state[state_y_idx]};

                i++;
            }

            return result;
        }
#endif

        // Private member functions of the controller
        void prefix_scan(thrust::device_ptr<float> normal) {
            auto actions = thrust::device_pointer_cast((DeviceAction*)normal.get());
            auto keys = thrust::make_transform_iterator(thrust::make_counting_iterator(0), DivBy<num_timesteps> {});

            thrust::inclusive_scan_by_key(keys, keys + num_samples * num_timesteps,
                                          actions, actions,
                                          Equal<size_t> {},
                                          AddActions {});
        }

        void MppiController_Impl::generate_brownians() {
            // generate normals, put it in device memory pointed to by m_action_trajectories
            // .data().get() returns the raw pointer from a device vector
            CURAND_CALL(hiprandGenerateNormal(m_rng, m_action_trajectories.data().get(), num_action_trajectories, 0, 1));

            // make the normals brownian
            thrust::counting_iterator<size_t> indices {0};
            thrust::for_each(indices, indices + num_action_trajectories, TransformStdNormal {m_action_trajectories.data()});
            prefix_scan(m_action_trajectories.data());
        }


        void MppiController_Impl::generate_log_probability_density() {
            
            // Calculates Log probability density
            thrust::counting_iterator<size_t> indices {0};
            thrust::for_each(
                indices, indices + num_samples * num_timesteps,
                LogProbabilityDensity {m_action_trajectories.data(), m_log_prob_densities.data()}
            );
        }


        thrust::device_vector<DeviceAction> MppiController_Impl::reduce_actions() {
            // averaged_actions is where the weighted averages are stored
            // initialize it to 0 
            thrust::device_vector<DeviceAction> averaged_actions (num_timesteps);
            thrust::device_vector<ActionWeightTuple> averaged_awts(num_timesteps);
            thrust::device_vector<uint32_t> keys_out (num_timesteps);
            thrust::counting_iterator<uint32_t> indices {0};
            auto keys = thrust::make_transform_iterator(indices, DivBy<num_samples> {});

            // for_each applies the ReduceTimestep functor to every idx in the range [0, num_timesteps)
            thrust::reduce_by_key(
                keys, keys + num_samples * num_timesteps, m_action_weight_tuples.begin(),
                keys_out.begin(), averaged_awts.begin()
            );

            thrust::transform(
                averaged_awts.begin(), averaged_awts.end(), averaged_actions.begin(),
                ActionWeightTupleToAction {}
            );

            return averaged_actions;
        }

        void MppiController_Impl::populate_cost() {
            thrust::counting_iterator<uint32_t> indices {0};

            PopulateCost populate_cost {
                m_action_trajectories.data(),
                m_action_trajectories.data(),
#ifdef DISPLAY
                m_state_trajectories.data(),
#endif
                m_cost_to_gos.data(),
                m_log_prob_densities.data(), 
                m_last_action_trajectory.data(),
                m_last_action
            };

            thrust::for_each(indices, indices + num_samples, populate_cost);
        }

        void MppiController_Impl::generate_action_weight_tuples() {
            thrust::counting_iterator<uint32_t> indices {0};

            IndexToActionWeightTuple transform_to_tuple {
                m_action_weight_tuples.data().get(),
                m_action_trajectories.data().get(),
                m_cost_to_gos.data().get(),
                m_log_prob_densities.data().get()
            };

            thrust::for_each(indices, indices + num_timesteps * num_samples, transform_to_tuple);
        }
    }
}