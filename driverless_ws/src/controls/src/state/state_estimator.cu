#include "hip/hip_runtime.h"
/**
 * To-Do list:
 * - Add drawing of small triangles corresponding to cones
 * - Automatic zoom-out/pan
 * - car boundary
 * 
 */


#ifndef GLM_FORCE_QUAT_DATA_WXYZ
#define GLM_FORCE_QUAT_DATA_WXYZ
#endif

#include <utils/cuda_utils.cuh>
#include <utils/gl_utils.hpp>
#include <cuda_globals/cuda_globals.cuh>
#include <glm/glm.hpp>
#include <glm/gtc/quaternion.hpp>
#include <cuda_constants.cuh>
#include <cmath>
#include <cuda_gl_interop.h>


#include "state_estimator.cuh"
#include "state_estimator.hpp"

#include <iosfwd>
#include <vector>
#include <sstream>
#include <rclcpp/rclcpp.hpp>
#include <glm/common.hpp>
#include <mppi/functors.cuh>
#include <SDL2/SDL_video.h>

#include <midline/svm_conv.hpp>
#include <utils/ros_utils.hpp>

namespace {
    // Helper function for timing with optional GPU synchronization
    template<bool EnableSync>
    std::chrono::time_point<std::chrono::high_resolution_clock> sync_now() {
        // if constexpr (EnableSync) {
        //     glFinish(); // Ensure GPU has completed all work
        // }
        return std::chrono::high_resolution_clock::now();
    }
    
    // Helper function to log timings if enabled
    template<bool EnableLog>
    void log_timings(
        rclcpp::Logger& logger,
        float total_ms,
        float gl_context_ms,
        float tex_info_ms,
        float buffer_cones_ms,
        float buffer_spline_ms,
        float unmap_ms,
        float fake_track_ms,
        float curv_frame_ms,
        float map_ms,
        float sync_state_ms,
        float sync_tex_ms,
        float display_ms,
        float unbind_ms) {
        
        if constexpr (EnableLog) {
            // Note: When EnableLog==true with glFinish calls, these timings include GPU execution time
            RCLCPP_INFO(logger, 
                "render_and_sync timing (ms): Total=%.2f, GL context=%.2f, Tex info=%.2f, "
                "Buffer cones=%.2f, Buffer spline=%.2f, Unmap=%.2f, Fake track=%.2f, "
                "Curv frame=%.2f, Map=%.2f, Sync state=%.2f, Sync tex=%.2f, Display=%.2f, Unbind=%.2f",
                total_ms, gl_context_ms, tex_info_ms,
                buffer_cones_ms, buffer_spline_ms, unmap_ms,
                fake_track_ms, curv_frame_ms, map_ms,
                sync_state_ms, sync_tex_ms, display_ms, unbind_ms);
            
            // Log sum of component times to verify accounting
            float sum = gl_context_ms + tex_info_ms + buffer_cones_ms + buffer_spline_ms + 
                      unmap_ms + fake_track_ms + curv_frame_ms + map_ms + 
                      sync_state_ms + sync_tex_ms + unbind_ms;
            
            RCLCPP_INFO(logger, "Sum of component times: %.2f ms (%.2f%% of total)",
                       sum, (sum / total_ms) * 100.0f);
        }
    }
}

namespace controls {
    namespace state {
 

        // State Estimator

        std::shared_ptr<StateEstimator> StateEstimator::create(std::mutex& mutex, LoggerFunc logger) {
            return std::make_shared<StateEstimator_Impl>(mutex, logger);
        }

        // StateEstimator_Impl helpers


        /**
         * @brief GPU shader code. In a string because historically, shaders are JIT compiled.
         * Runs in parallel for every vertex in the VBO, similar to a functor.
         * Layouts initially specified in gen_gl_path().
         *
         * Transforms from vertex IRL position to clip space (rendering coordinate frame) using scale
         * and center. For now this transformation is unneeded. However, suppose with SLAM, (x,y) from
         * path planning are from a stationary world perspective, but we want to render to the car's vicinity.
         *
         * @note: the z coordinate in gl_Position is how we exploit depth testing to break ties between multiple
         * overlapping triangles.
         * note that i_world_pos is a vec2 so the dimensions in gl_Position check out (shader languages are built for vector math)
         * uniform is similar to __constant__ in CUDA
         * @note: far_frustrum is for better precision since only relative ordering matters
         * @note: we don't use the 4th coordinate of gl_Position, but it is needed. Look up homogenous coordinates.
         *
         * @param[in] i_world_pos x, y from path planning
         * @param[in] i_curv_pose progress, offset, heading
         * @param[out] o_curv_pose same as i_curv_pose, passed along
         * @return gl_Position
         */
        constexpr const char* vertex_source_fake_track = R"(
            #version 330 core
            #extension GL_ARB_explicit_uniform_location : enable

            layout (location = 0) in vec2 i_world_pos;
            layout (location = 1) in vec3 i_curv_pose;

            out vec3 o_curv_pose;

            layout (location = 0) uniform float scale;
            layout (location = 1) uniform vec2 center;

            const float far_frustum = 10.0f;

            void main() {
                gl_Position = vec4(scale * (i_world_pos - center), abs(i_curv_pose.y) / far_frustum, 1.0);
                o_curv_pose = i_curv_pose;
            }
        )";

        /**
         * @brief GPU shader code. "fragment shader"
         * Runs in parallel for every pixel in the triangles.
         * We only manually calculated o_curv_pose for the vertices,
         * interpolation for the pixels in between happens here automatically.
         * 4th color (1.0f) represents in bounds, compared to background which has -1.0 representing OOB
         * @return FragColor The color of each foreground pixel
         */
        constexpr const char* fragment_source_fake_track = R"(
            #version 330 core

            in vec3 o_curv_pose;

            out vec4 FragColor;

            void main() {
                FragColor = vec4(o_curv_pose, 1.0f);
            }
        )";

        // TODO: I think abs(i_curv_pose.y) is not needed, and hence i_curv_pose is not needed, because glLineStrip
        // means there will be no overlapping triangles
        // besides cones don't have offset information
        constexpr const char *vertex_source = R"(
            #version 330 core
            #extension GL_ARB_explicit_uniform_location : enable

            layout (location = 0) in vec2 i_world_pos;
            layout (location = 1) in vec3 i_curv_pose;

            out vec2 o_world_pose;

            layout (location = 0) uniform float scale;
            layout (location = 1) uniform vec2 center;

            const float far_frustum = 10.0f;

            void main() {
                o_world_pose = scale * (i_world_pos - center);
                gl_Position = vec4(scale * (i_world_pos - center), abs(i_curv_pose.y) / far_frustum, 1.0);
            }
        )";

        constexpr const char *fragment_source = R"(
            #version 330 core

            in vec2 o_world_pose;

            out vec4 FragColor;

            uniform sampler2D fake_track_texture;

            void main() {
                FragColor = texture(fake_track_texture, o_world_pose / 2.0 + 0.5); // convert from normalized device coordinates to texture coordinates
            }
        )";

        // methods

        // Constructor
        StateEstimator_Impl::StateEstimator_Impl(std::mutex& mutex, LoggerFunc logger)
            : m_mutex {mutex}, m_logger {logger}, m_logger_obj {rclcpp::get_logger("")} {
            std::lock_guard<std::mutex> guard {mutex};

            m_logger("initializing state estimator");
#ifdef DISPLAY
            m_gl_window = utils::create_sdl2_gl_window(
                "Spline Frame Lookup", curv_frame_lookup_tex_width, curv_frame_lookup_tex_width,
                0, &m_gl_context
            );
#else
            // dummy window to create opengl context for curv frame buffer
            m_gl_window = utils::create_sdl2_gl_window(
                "Spline Frame Lookup Dummy", 1, 1,
                SDL_WINDOW_HIDDEN, &m_gl_context
            );
#endif

            m_logger("making state estimator gl context current");
            utils::make_gl_current_or_except(m_gl_window, m_gl_context);

            m_logger("compiling state estimator shaders");
            
            m_fake_track_shader_program = utils::compile_shader(vertex_source_fake_track, fragment_source_fake_track);
            m_gl_path_shader = utils::compile_shader(vertex_source, fragment_source);

            m_logger("setting state estimator gl properties");

            glViewport(0, 0, curv_frame_lookup_tex_width, curv_frame_lookup_tex_width);

            m_logger("generating state estimator gl buffers");
            gen_curv_frame_lookup_framebuffer();
            gen_gl_path(m_gl_path);
            gen_fake_track();

            glFinish();
            utils::make_gl_current_or_except(m_gl_window, nullptr);
            m_logger("finished state estimator initialization");
            SDL_GLContext curr_context = SDL_GL_GetCurrentContext();
            SDL_Window* curr_window = SDL_GL_GetCurrentWindow();
            // RCLCPP_INFO(m_logger_obj, "After constructor: window: %p, context %p", curr_window, curr_context);


        }

        void StateEstimator_Impl::gen_curv_frame_lookup_framebuffer() {
            glGenFramebuffers(1, &m_curv_frame_lookup_fbo);
            glBindFramebuffer(GL_FRAMEBUFFER, m_curv_frame_lookup_fbo);

            glGenRenderbuffers(1, &m_curv_frame_lookup_rbo);
            glBindRenderbuffer(GL_RENDERBUFFER, m_curv_frame_lookup_rbo);
            glRenderbufferStorage(GL_RENDERBUFFER, GL_RGBA32F, curv_frame_lookup_tex_width, curv_frame_lookup_tex_width);
            glFramebufferRenderbuffer(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_RENDERBUFFER, m_curv_frame_lookup_rbo);

            if (glCheckFramebufferStatus(GL_FRAMEBUFFER) != GL_FRAMEBUFFER_COMPLETE) {
                throw std::runtime_error("Framebuffer is not complete");
            }
            // reset framebuffer to default
            glBindFramebuffer(GL_FRAMEBUFFER, 0);
        }

        void StateEstimator_Impl::gen_fake_track() {
            // generate the framebuffer for the fake track
            glGenFramebuffers(1, &m_fake_track_fbo);
            glBindFramebuffer(GL_FRAMEBUFFER, m_fake_track_fbo);

            // generate texture
            glGenTextures(1, &m_fake_track_texture_color);
            glBindTexture(GL_TEXTURE_2D, m_fake_track_texture_color);

            glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, curv_frame_lookup_tex_width, curv_frame_lookup_tex_width, 0, GL_RGBA, GL_FLOAT, NULL);
            glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
            glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
            glBindTexture(GL_TEXTURE_2D, 0);

            // attach it to currently bound framebuffer object
            glFramebufferTexture2D(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_TEXTURE_2D, m_fake_track_texture_color, 0); 
            
            GLuint depth_rbo;

            glGenRenderbuffers(1, &depth_rbo);
            glBindRenderbuffer(GL_RENDERBUFFER, depth_rbo);
            glRenderbufferStorage(GL_RENDERBUFFER, GL_DEPTH_COMPONENT32, curv_frame_lookup_tex_width,  curv_frame_lookup_tex_width);
            glFramebufferRenderbuffer(GL_FRAMEBUFFER, GL_DEPTH_ATTACHMENT, GL_RENDERBUFFER, depth_rbo);

            if (glCheckFramebufferStatus(GL_FRAMEBUFFER) != GL_FRAMEBUFFER_COMPLETE) {
                throw std::runtime_error("Fake track framebuffer is not complete");
            }
            // reset framebuffer to default
            glBindFramebuffer(GL_FRAMEBUFFER, 0);

            gen_gl_path(m_fake_track_path);
        }

        StateEstimator_Impl::~StateEstimator_Impl() {
            // utils::sync_gl_and_unbind_context(m_gl_window);
            SDL_QuitSubSystem(SDL_INIT_VIDEO);
        }

        std::vector<glm::fvec2> process_ros_points(std::vector<geometry_msgs::msg::Point,
                                                               std::allocator<geometry_msgs::msg::Point>>
                                                       points)
        {
            std::vector<glm::fvec2> processed_points;
            processed_points.reserve(points.size());
            for (const auto &point : points)
            {
                float cone_y = static_cast<float>(point.y);
                float cone_x = static_cast<float>(point.x);
                processed_points.push_back(
                    glm::fvec2(cone_x, cone_y));
            }
            paranoid_assert(processed_points.size() == points.size());
            return processed_points;
        }

        void StateEstimator_Impl::on_spline(const SplineMsg& spline_msg) {
            std::lock_guard<std::mutex> guard {m_mutex};

            m_logger("beginning state estimator spline processing");

            paranoid_assert(spline_msg.frames.size() > 0);

            if constexpr (ingest_midline) {
                m_spline_frames = process_ros_points(spline_msg.frames);
            }

            m_logger("finished state estimator spline processing");
        }

        void StateEstimator_Impl::on_quat(const QuatMsg& quat_msg) {
            std::lock_guard<std::mutex> guard {m_mutex};
            const float yaw = quat_msg_to_yaw(quat_msg);
            m_state_projector.record_yaw(yaw, quat_msg.header.stamp);
        }

        float StateEstimator_Impl::on_cone(const ConeMsg& cone_msg) {
            std::lock_guard<std::mutex> guard {m_mutex};

            paranoid_assert(cone_msg.blue_cones.size() > 0);
            paranoid_assert(cone_msg.yellow_cones.size() > 0);

            m_logger("beginning state estimator cone processing");

            m_left_cone_points.clear();
            m_right_cone_points.clear();

            m_left_cone_points = process_ros_points(cone_msg.blue_cones);
            m_right_cone_points = process_ros_points(cone_msg.yellow_cones);

            float svm_time = 0.0f;

            if constexpr (!ingest_midline) {

                midline::Cones cones;
                for (const auto& cone : m_left_cone_points) {
                    cones.addBlueCone(cone.x, cone.y, 0);
                }
                for (const auto& cone : m_right_cone_points) {
                    cones.addYellowCone(cone.x, cone.y, 0);
                }

                // // TODO: convert this to using std::transform
                auto svm_start = std::chrono::high_resolution_clock::now();            
                auto spline_frames = midline::svm_fast::cones_to_midline(cones);
                // auto spline_frames = midline::svm_slow::cones_to_midline(cones);
                auto svm_end = std::chrono::high_resolution_clock::now();
                svm_time = std::chrono::duration_cast<std::chrono::milliseconds>(svm_end - svm_start).count();
                m_spline_frames.clear();
                for (const auto& frame : spline_frames) {
                    paranoid_assert(!isnan(frame.first) && !isnan(frame.second));
                    m_spline_frames.emplace_back(frame.first, frame.second);
                }   

            }


#ifdef DISPLAY
            m_all_left_cone_points.clear();
            m_all_right_cone_points.clear();

            m_all_left_cone_points = process_ros_points(cone_msg.orange_cones);
            m_all_right_cone_points = process_ros_points(cone_msg.unknown_color_cones);
            m_raceline_points = process_ros_points(cone_msg.big_orange_cones);
#endif

            if constexpr (reset_pose_on_cone) {
                switch (state_projection_mode) {
                    case StateProjectionMode::MODEL_MULTISET: {
                        m_state_projector.record_pose(0, 0, M_PI_2, cone_msg.header.stamp);
                    }
                    break;
                    default:
                    break;
                }
            }
            
            m_logger("finished state estimator cone processing");
            return svm_time;
        }

        void StateEstimator_Impl::on_twist(const TwistMsg &twist_msg, const rclcpp::Time &time) {
            // TODO: whats up with all these mutexes
            std::lock_guard<std::mutex> guard {m_mutex};

            const float speed = twist_msg_to_speed(twist_msg);

            m_state_projector.record_speed(speed, time);

        }

        void StateEstimator_Impl::on_pose(const PoseMsg &pose_msg) {
            std::lock_guard<std::mutex> guard {m_mutex};
            m_state_projector.record_pose(
                pose_msg.pose.position.x, pose_msg.pose.position.y, pose_msg.pose.orientation.z,
                pose_msg.header.stamp);
        }


        void StateEstimator_Impl::record_control_action(const Action& action, const rclcpp::Time& time) {
            std::lock_guard<std::mutex> guard {m_mutex};
            // TODO: put this under a constexpr switch flag, now we always record for debugging purposes
            m_state_projector.record_action(action, rclcpp::Time{
                                                        time.nanoseconds() + static_cast<int64_t>(approx_propogation_delay * 1e9f),
                                                        default_clock_type});

        }

        void StateEstimator_Impl::render_and_sync(State state) {
            std::lock_guard<std::mutex> guard {m_mutex};
            
            // Timing variables
            auto start_time = sync_now<log_render_and_sync_timing>();
            auto last_time = start_time;
            std::chrono::duration<float, std::milli> gl_context_time, tex_info_time, 
                buffer_cones_time, buffer_spline_time, unmap_time, 
                fake_track_time, curv_frame_time, map_time, 
                sync_state_time, sync_tex_time, display_time, unbind_time, total_time;
            

            SDL_GLContext log_context;
            SDL_Window* log_window;
            if constexpr (log_render_and_sync_timing) {
                log_context = SDL_GL_GetCurrentContext();
                log_window = SDL_GL_GetCurrentWindow();
                RCLCPP_INFO(m_logger_obj, "Before start of render and sync: window: %p, context %p", log_window, log_context);
            }



            // enable openGL
            utils::make_gl_current_or_except(m_gl_window, m_gl_context);
            if constexpr (log_render_and_sync_timing) {
                RCLCPP_INFO(m_logger_obj, "Call to make gl current or except: window: %p, context %p", m_gl_window, m_gl_context);
            }
            

            // // exclusively for logging
            if constexpr (log_render_and_sync_timing) {
                log_context = SDL_GL_GetCurrentContext();
                log_window = SDL_GL_GetCurrentWindow();
                RCLCPP_INFO(m_logger_obj, "After make gl current in render and sync - window: %p, context %p", log_window, log_context);
            }


            auto current_time = sync_now<log_render_and_sync_timing>();
            gl_context_time = current_time - last_time;
            last_time = current_time;

            m_logger("generating spline frame lookup texture info...");
            gen_tex_info({state[state_x_idx], state[state_y_idx]});
            current_time = sync_now<log_render_and_sync_timing>();
            tex_info_time = current_time - last_time;
            last_time = current_time;

            m_logger("filling OpenGL buffers...");
            // takes car position, places them in the vertices
            fill_path_buffers_cones();
            current_time = sync_now<log_render_and_sync_timing>();
            buffer_cones_time = current_time - last_time;
            last_time = current_time;
            
            fill_path_buffers_spline();
            current_time = sync_now<log_render_and_sync_timing>();
            buffer_spline_time = current_time - last_time;
            last_time = current_time;

            m_logger("unmapping CUDA curv frame lookup texture for OpenGL rendering");
            unmap_curv_frame_lookup();
            current_time = sync_now<log_render_and_sync_timing>();
            unmap_time = current_time - last_time;
            last_time = current_time;

            // render the lookup table
            m_logger("rendering curv frame lookup table...");
            render_fake_track();
            current_time = sync_now<log_render_and_sync_timing>();
            fake_track_time = current_time - last_time;
            last_time = current_time;
            
            if (!m_follow_midline_only) {
                render_curv_frame_lookup();
                current_time = sync_now<log_render_and_sync_timing>();
                curv_frame_time = current_time - last_time;
                last_time = current_time;
            }

            m_logger("mapping OpenGL curv frame texture back to CUDA");
            map_curv_frame_lookup();
            current_time = sync_now<log_render_and_sync_timing>();
            map_time = current_time - last_time;
            last_time = current_time;

            m_logger("syncing world state to device");
            CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cuda_globals::curr_state), state.data(), state_dims * sizeof(float)));
            current_time = sync_now<log_render_and_sync_timing>();
            sync_state_time = current_time - last_time;
            last_time = current_time;

            m_logger("syncing spline frame lookup texture info to device");
            sync_tex_info();
            current_time = sync_now<log_render_and_sync_timing>();
            sync_tex_time = current_time - last_time;
            last_time = current_time;


#ifdef DISPLAY
            m_last_offset_image.pixels = std::vector<float>(4 * curv_frame_lookup_tex_width * curv_frame_lookup_tex_width);
            glBindFramebuffer(GL_READ_FRAMEBUFFER, m_curv_frame_lookup_fbo);
            glReadPixels(
                0, 0, curv_frame_lookup_tex_width, curv_frame_lookup_tex_width,
                GL_RGBA, GL_FLOAT,
                m_last_offset_image.pixels.data()
            );

            m_last_offset_image.pix_width = curv_frame_lookup_tex_width;
            m_last_offset_image.pix_height = curv_frame_lookup_tex_width;
            m_last_offset_image.center = {m_curv_frame_lookup_tex_info.xcenter, m_curv_frame_lookup_tex_info.ycenter};
            m_last_offset_image.world_width = m_curv_frame_lookup_tex_info.width;
#endif
            current_time = sync_now<log_render_and_sync_timing>();
            display_time = current_time - last_time;
            last_time = current_time;

            utils::sync_gl_and_unbind_context(m_gl_window);
            if constexpr (log_render_and_sync_timing) {
                log_context = SDL_GL_GetCurrentContext();
                log_window = SDL_GL_GetCurrentWindow();
                RCLCPP_INFO(m_logger_obj, "What happens after unbinding?: window: %p, context %p", log_window, log_context);
            }

            current_time = sync_now<log_render_and_sync_timing>();
            unbind_time = current_time - last_time;
            
            // Total time
            total_time = current_time - start_time;
            
            // Log timing information using our helper function
            log_timings<log_render_and_sync_timing>(
                m_logger_obj,
                total_time.count(),
                gl_context_time.count(),
                tex_info_time.count(),
                buffer_cones_time.count(),
                buffer_spline_time.count(),
                unmap_time.count(),
                fake_track_time.count(),
                curv_frame_time.count(),
                map_time.count(),
                sync_state_time.count(),
                sync_tex_time.count(),
                display_time.count(),
                unbind_time.count()
            );
        }

        std::optional<State> StateEstimator_Impl::project_state(const rclcpp::Time& time) {
            std::lock_guard<std::mutex> guard {m_mutex};
            auto state = m_state_projector.project(
                rclcpp::Time{
                    time.nanoseconds() + static_cast<int64_t>((approx_propogation_delay + approx_mppi_time) * 1e9f),
                    default_clock_type},
                m_logger);
                 
            return state;
        }


        bool StateEstimator_Impl::is_ready() {
            std::lock_guard<std::mutex> guard {m_mutex};

            return m_state_projector.is_ready();
        }

        void StateEstimator_Impl::set_logger(LoggerFunc logger) {
            std::lock_guard<std::mutex> guard {m_mutex};

            m_logger = logger;
        }

        void StateEstimator_Impl::set_logger_obj(rclcpp::Logger logger)
        {
            std::lock_guard<std::mutex> guard {m_mutex};
            m_logger_obj = logger;
        }

        std::vector<glm::fvec2> StateEstimator_Impl::get_spline_frames()
        {
            std::lock_guard<std::mutex> guard{m_mutex};

            std::vector<glm::fvec2> res(m_spline_frames.size());
            for (size_t i = 0; i < m_spline_frames.size(); i++)
            {
                res[i] = {m_spline_frames[i].x, m_spline_frames[i].y};
            }
            return res;
        }

#ifdef DISPLAY
        std::vector<glm::fvec2> StateEstimator_Impl::get_all_left_cone_points() {
            std::lock_guard<std::mutex> guard {m_mutex};        
            return m_all_left_cone_points;
        }

        std::vector<glm::fvec2> StateEstimator_Impl::get_all_right_cone_points() {
            std::lock_guard<std::mutex> guard {m_mutex};
            
            return m_all_right_cone_points;
        }

        std::vector<glm::fvec2> StateEstimator_Impl::get_left_cone_points() {
            std::lock_guard<std::mutex> guard {m_mutex};        
            return m_left_cone_points;
        }

        std::vector<glm::fvec2> StateEstimator_Impl::get_right_cone_points() {
            std::lock_guard<std::mutex> guard {m_mutex};
            
            return m_right_cone_points;
        }

        // *****REVIEW: not be needed for display
        std::vector<glm::fvec2> StateEstimator_Impl::get_raceline_points(){
            std::lock_guard<std::mutex> guard {m_mutex};
            std::stringstream ss;

            ss << "Raceline points size: " << m_raceline_points.size() << "\n";
            for (size_t i = 0; i < m_raceline_points.size(); i++)
            {
                ss << "Index: " << i << " Point x: " << m_raceline_points[i].x << "Point y: " << m_raceline_points[i].y << "\n";
            }

            return m_raceline_points;
        }


        std::vector<float> StateEstimator_Impl::get_vertices() {
            std::lock_guard<std::mutex> guard {m_mutex};

            return m_vertices;
        }

        // std::vector<GLuint> StateEstimator_Impl::get_indices() {
        //     std::lock_guard<std::mutex> guard {m_mutex};

        //     return m_indices;


        OffsetImage StateEstimator_Impl::get_offset_pixels() {
            std::lock_guard<std::mutex> guard {m_mutex};
            return m_last_offset_image;
        }
#endif

        void StateEstimator_Impl::sync_tex_info() {
            CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(
                cuda_globals::curv_frame_lookup_tex_info), &m_curv_frame_lookup_tex_info, sizeof(cuda_globals::CurvFrameLookupTexInfo)
            ));
        }

        void StateEstimator_Impl::gen_tex_info(glm::fvec2 car_pos) {
            float xmin = car_pos.x;
            float ymin = car_pos.y;
            float xmax = car_pos.x;
            float ymax = car_pos.y;

            for (const glm::fvec2 frame : m_spline_frames) {
                xmin = std::min(xmin, frame.x);
                xmax = std::max(xmax, frame.x);
                ymin = std::min(ymin, frame.y);
                ymax = std::max(ymax, frame.y);
            }

            m_curv_frame_lookup_tex_info.xcenter = (xmax + xmin) / 2;
            m_curv_frame_lookup_tex_info.ycenter = (ymax + ymin) / 2;
            m_curv_frame_lookup_tex_info.width = std::max(xmax - xmin, ymax - ymin) + car_padding * 2;
        }

        void StateEstimator_Impl::render_fake_track() {
            glBindFramebuffer(GL_FRAMEBUFFER, m_fake_track_fbo);

            if (m_follow_midline_only) {
                // ^ Replaces the texture with the render buffer (the final target)
                // Explanation: If we are only following the midline, we don't need track bounds, so we can skip the second rendering step
                glFramebufferRenderbuffer(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_RENDERBUFFER, m_curv_frame_lookup_rbo);
            }

            glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
            glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
            glEnable(GL_DEPTH_TEST);
            glDepthFunc(GL_LESS);

            // use a shader program
            glUseProgram(m_fake_track_shader_program);
            // set the relevant scale and center uniforms (constants) in the shader program
            glUniform1f(shader_scale_loc, 2.0f / m_curv_frame_lookup_tex_info.width);
            glUniform2f(shader_center_loc, m_curv_frame_lookup_tex_info.xcenter, m_curv_frame_lookup_tex_info.ycenter);

            glBindVertexArray(m_fake_track_path.vao);
            glDrawElements(GL_TRIANGLES, (m_spline_frames.size() * 6 - 2) * 3, GL_UNSIGNED_INT, nullptr);

#ifdef DISPLAY
            glBindFramebuffer(GL_READ_FRAMEBUFFER, m_fake_track_fbo);
            glBindFramebuffer(GL_DRAW_FRAMEBUFFER, 0);
            glBlitFramebuffer(
                0, 0, curv_frame_lookup_tex_width, curv_frame_lookup_tex_width,
                0, 0, curv_frame_lookup_tex_width, curv_frame_lookup_tex_width,
                GL_COLOR_BUFFER_BIT, GL_NEAREST);

            SDL_GL_SwapWindow(m_gl_window);
#endif
        }

        void StateEstimator_Impl::render_curv_frame_lookup() {
            // tells OpenGL: this is where I want to render to
            glBindFramebuffer(GL_FRAMEBUFFER, m_curv_frame_lookup_fbo);

            // set the background color, clears the depth buffer
            // (technically 2 rendering passes are done - color and depth)
            glClearColor(0.0f, 0.0f, 0.0f, 0.0f);
            glClear(GL_COLOR_BUFFER_BIT); // TODO: remove depth buffer
            // use a shader program
            glUseProgram(m_gl_path_shader);
            // set the relevant scale and center uniforms (constants) in the shader program
            glUniform1f(shader_scale_loc, 2.0f / m_curv_frame_lookup_tex_info.width);
            glUniform2f(shader_center_loc, m_curv_frame_lookup_tex_info.xcenter, m_curv_frame_lookup_tex_info.ycenter);

            glBindVertexArray(m_gl_path.vao);
            glBindTexture(GL_TEXTURE_2D, m_fake_track_texture_color);
            glDrawArrays(GL_TRIANGLES, 0, m_num_triangles*3);


        }

        // Whole lotta CUDA nonsense. Tread lightly.
        void StateEstimator_Impl::map_curv_frame_lookup() {
            CUDA_CALL(hipGraphicsGLRegisterImage(&m_curv_frame_lookup_rsc, m_curv_frame_lookup_rbo, GL_RENDERBUFFER, hipGraphicsRegisterFlagsNone));

            if (!m_curv_frame_lookup_mapped) {
                m_curv_frame_lookup_mapped = true;

                CUDA_CALL(hipGraphicsMapResources(1, &m_curv_frame_lookup_rsc));
            }

            hipResourceDesc img_rsc_desc {};
            img_rsc_desc.resType = hipResourceTypeMipmappedArray;
            CUDA_CALL(cudaGraphicsResourceGetMappedMipmappedArray(&img_rsc_desc.res.mipmap.mipmap, m_curv_frame_lookup_rsc));

            hipTextureDesc img_tex_desc {};
            img_tex_desc.addressMode[0] = hipAddressModeClamp;
            img_tex_desc.addressMode[1] = hipAddressModeClamp;
            img_tex_desc.filterMode = hipFilterModeLinear;
            img_tex_desc.readMode = hipReadModeElementType;
            img_tex_desc.normalizedCoords = true;

            hipTextureObject_t tex;
            CUDA_CALL(hipCreateTextureObject(&tex, &img_rsc_desc, &img_tex_desc, nullptr));
            CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(
                cuda_globals::curv_frame_lookup_tex), &tex, sizeof(hipTextureObject_t)
            ));
        }

        /// you can't render to something when it is mapped. have to unmap before rendering
        void StateEstimator_Impl::unmap_curv_frame_lookup() {
            if (!m_curv_frame_lookup_mapped)
                return;

            CUDA_CALL(hipGraphicsUnmapResources(1, &m_curv_frame_lookup_rsc));
            m_curv_frame_lookup_mapped = false;
        }


        // bind stuff to flags, act on flags, state machine

        /**
         * Creates the buffers to be used, as well as the descriptions of how the buffers are laid out.
         * @brief Creates the names for the vao, vbo and ebo.
         * Specifies how the vbo should be laid out, stores this in the vao.
         * Lastly, binds to the ebo.
         */
        void StateEstimator_Impl::gen_gl_path(utils::GLObj &gl_path) {
            // Generates the vao, vbo and ebo to be bound later.
            glGenVertexArrays(1, &gl_path.vao);
            glGenBuffers(1, &gl_path.vbo);
            glGenBuffers(1, &gl_path.ebo);

            glBindVertexArray(gl_path.vao);
            // OpenGL is a state machine, binding here means any relevant function call on a buffer will be on
            // m_gl_path.vbo until it is unbound.
            glBindBuffer(GL_ARRAY_BUFFER, gl_path.vbo);
            // Specifies the layout of the vertex buffer object. world_pos (2) and curv_pose (3).
            glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)0);
            glEnableVertexAttribArray(0);
            glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)(2 * sizeof(float)));
            glEnableVertexAttribArray(1);
            // vbo unbound here, ebo bound in its place.
            glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, gl_path.ebo);

            // vao is unbound, saving the settings.
            glBindVertexArray(0);
        }

        // Track bounds version
        void StateEstimator_Impl::fill_path_buffers_cones(){
            const size_t num_left_cones = m_left_cone_points.size();
            const size_t num_right_cones = m_right_cone_points.size();
            m_num_triangles = 0;
            std::vector<StateEstimator_Impl::Vertex> indices;
            std::vector<StateEstimator_Impl::Vertex> vertices;
            //vertices.reserve(num_left_cones + num_right_cones);
            for (size_t i = 0; i < num_left_cones; ++i) {
                    glm::fvec2 l1 = m_left_cone_points.at(i);
                    vertices.push_back({{l1.x, l1.y}, {0.0f, 0.3f, 0.0f}});
            }
            for (size_t i = 0; i < num_right_cones; ++i) {
                    glm::fvec2 r1 = m_right_cone_points.at(i);
                    vertices.push_back({{r1.x, r1.y}, {0.0f, 0.3f, 0.0f}});
            }
            float distance2;
            std::vector<GLuint> temp;
            for(size_t i = 0; i < num_left_cones; ++i){
                glm::fvec2 l1 = m_left_cone_points.at(i);
                distance2 = 0;
                temp.clear();
                for(size_t j = 0; j < num_right_cones; ++j){
                    glm::fvec2 r1 = m_right_cone_points.at(j);
                    distance2 = (l1.x - r1.x)*(l1.x - r1.x) + (l1.y - r1.y)*(l1.y - r1.y);
                    if(distance2 < triangle_threshold_squared)
                    {
                        temp.push_back(j);
                    }
                }
                if(temp.size() > 1){
                    for(size_t k = 0; k < temp.size()-1; ++k){
                        indices.push_back(vertices.at(i));
                        indices.push_back(vertices.at(temp.at(k)+ num_left_cones));
                        indices.push_back(vertices.at(temp.at(k+1) + num_left_cones));
                        m_num_triangles += 1;
                    }
                }
            }
            for(size_t i = 0; i < num_right_cones; ++i){
                glm::fvec2 r1 = m_right_cone_points.at(i);
                distance2 = 0;
                temp.clear();
                for(size_t j = 0; j < num_left_cones; j++){
                    glm::fvec2 l1 = m_left_cone_points.at(j);
                    distance2 = (l1.x - r1.x)*(l1.x - r1.x) + (l1.y - r1.y)*(l1.y - r1.y);
                    if(distance2 < triangle_threshold_squared)
                    {
                        temp.push_back(j);
                    }
                }
                if(temp.size() > 1){
                    for(size_t k = 0; k < temp.size()-1; ++k){
                        indices.push_back(vertices.at(i + num_left_cones));
                        indices.push_back(vertices.at(temp.at(k)));
                        indices.push_back(vertices.at(temp.at(k+1)));
                        m_num_triangles += 1;
                    }
                }

            }
            // TODO: decide whether we are going to keep using glDrawArrays or try to fix glDrawElements

            std::stringstream ss;
            ss << "Start of right at: " << num_left_cones;
            for(size_t i = 0; i < indices.size(); i++){
                ss << "Index: " << i << " Point x: " << indices.at(i).world.x << "Point y: "<< indices.at(i).world.y << "\n";
            }
            // if(indices.size() > 2) {
            //     for(size_t i = 0; i < indices.size()-2; i += 3){
            //         ss << "Index: " << indices.at(i) << " 2: " << indices.at(i+1) << " 3: " << indices.at(i+2) <<"------";
            //     }
            // }
    
            RCLCPP_DEBUG(m_logger_obj, ss.str().c_str());
            glBindBuffer(GL_ARRAY_BUFFER, m_gl_path.vbo);
            glBufferData(GL_ARRAY_BUFFER, sizeof(StateEstimator_Impl::Vertex) * indices.size(), indices.data(), GL_DYNAMIC_DRAW);

            // glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, m_gl_path.ebo);
            // glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(GLuint) * indices.size(), indices.data(), GL_DYNAMIC_DRAW);

        }
        
        // Offset from spline version
        /**
         * Given spline frames, fills a framebuffer object with the fake track corresponding to the spline. Used for track progress.
         * The framebuffer object will be used as a texture object to be sampled from the state estimator.
         */
        void StateEstimator_Impl::fill_path_buffers_spline() {
            for (const auto& frame : m_spline_frames) {
                paranoid_assert(!isnan_vec(frame));
            }
            struct Vertex {
                struct {
                    float x;
                    float y;
                } world;

                struct {
                    float progress;
                    float offset;
                    float heading;
                } curv;
            };

            const float radius = fake_track_width * 0.5f;
            const size_t n = m_spline_frames.size();

            if (n < 2) {
                throw ControllerError("less than 2 spline frames! (bruh saket and/or saket)");
            }

            std::vector<Vertex> vertices;
            std::vector<GLuint> indices;

            float total_progress = 0;
            for (size_t i = 0; i < n - 1; i++) {
                glm::fvec2 p1 = m_spline_frames[i];
                glm::fvec2 p2 = m_spline_frames[i + 1];

                glm::fvec2 unit_vec = glm::length(p2 - p1) != 0 ? glm::normalize(p2 - p1) : glm::fvec2(0, 0);
                // This creates a longitudinal buffer at the start and the end of the spline for the fake track to be rendered
                if (i == 0) {
                    p1 = p1 - unit_vec * car_padding;
                } else if (i == n - 2) {
                    p2 = p2 + unit_vec * car_padding;
                }

                glm::fvec2 disp = p2 - p1;
                float new_progress = glm::length(disp); // TODO: figure out a way to normalize without some arbitrary magic number
                paranoid_assert(!isnan(new_progress));
                // 1. go through the vector and divide based on total progress
                // 2. set total progress to be a member variable, then use that as a uniform, thus passing it into the fragment shader
                float segment_heading = std::atan2(disp.y, disp.x);


                glm::fvec2 prev = i == 0 ? p1 : m_spline_frames[i - 1];
                float secant_heading = std::atan2(p2.y - prev.y, p2.x - prev.x);

                glm::fvec2 dir = glm::normalize(disp);
                glm::fvec2 normal = glm::fvec2(-dir.y, dir.x);

                glm::fvec2 low1 = p1 - normal * radius;
                glm::fvec2 low2 = p2 - normal * radius;
                glm::fvec2 high1 = p1 + normal * radius;
                glm::fvec2 high2 = p2 + normal * radius;

                if (i == 0)
                {
                    vertices.push_back({{p1.x, p1.y}, {total_progress, 0.0f, 0.0f}});
                }
                vertices.push_back({{p2.x, p2.y}, {total_progress + new_progress, 0.0f, 0.0f}});

                // I set offset to be 1.0 to prevent plateauing
                vertices.push_back({{low1.x, low1.y}, {total_progress, radius, 0.0f}});
                vertices.push_back({{low2.x, low2.y}, {total_progress + new_progress, radius, 0.0f}});
                vertices.push_back({{high1.x, high1.y}, {total_progress, radius, 1.0f}});
                vertices.push_back({{high2.x, high2.y}, {total_progress + new_progress, radius, 1.0f}});

                const GLuint p1i = i == 0 ? 0 : (i - 1) * 5 + 1;
                const GLuint p2i = i * 5 + 1;
                const GLuint l1i = i * 5 + 2;
                const GLuint l2i = i * 5 + 3;
                const GLuint h1i = i * 5 + 4;
                const GLuint h2i = i * 5 + 5;

                indices.push_back(p1i);
                indices.push_back(p2i);
                indices.push_back(h2i);

                indices.push_back(h1i);
                indices.push_back(p1i);
                indices.push_back(h2i);

                indices.push_back(l1i);
                indices.push_back(l2i);
                indices.push_back(p2i);

                indices.push_back(p1i);
                indices.push_back(l1i);
                indices.push_back(p2i);

                if (i > 0) {
                    const GLuint lpi = (i - 1) * 5 + 3;
                    const GLuint hpi = (i - 1) * 5 + 5;

                    indices.push_back(hpi);
                    indices.push_back(p1i);
                    indices.push_back(h1i);

                    indices.push_back(lpi);
                    indices.push_back(l1i);
                    indices.push_back(p1i);
                }

                total_progress += new_progress;
            }

            glBindBuffer(GL_ARRAY_BUFFER, m_fake_track_path.vbo);
            glBufferData(GL_ARRAY_BUFFER, sizeof(Vertex) * vertices.size(), vertices.data(), GL_DYNAMIC_DRAW);

            glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, m_fake_track_path.ebo);
            glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(GLuint) * indices.size(), indices.data(), GL_DYNAMIC_DRAW);
        }
    }
}

