#include "hip/hip_runtime.h"
#include <utils/cuda_utils.cuh>
#include <utils/gl_utils.hpp>
#include <cuda_globals/cuda_globals.cuh>
#include <glm/glm.hpp>
#include <glm/gtc/quaternion.hpp>
#include <cuda_constants.cuh>
#include <cmath>
#include <cuda_gl_interop.h>


#include "state_estimator.cuh"
#include "state_estimator.hpp"

#include <SDL2/SDL_video.h>


namespace controls {
    namespace state {

        std::shared_ptr<StateEstimator> StateEstimator::create(std::mutex& mutex) {
            return std::make_shared<StateEstimator_Impl>(mutex);
        }


        // StateEstimator_Impl helpers

        constexpr const char* vertex_source = R"(
            #version 330 core
            #extension GL_ARB_explicit_uniform_location : enable

            layout (location = 0) in vec2 i_world_pos;
            layout (location = 1) in vec3 i_curv_pose;

            out vec3 o_curv_pose;

            layout (location = 0) uniform float scale;
            layout (location = 1) uniform vec2 center;

            const float far_frustum = 10.0f;

            void main() {
                gl_Position = vec4(scale * (i_world_pos - center), abs(i_curv_pose.y) / far_frustum, 1.0);
                o_curv_pose = i_curv_pose;
            }
        )";

        constexpr const char* fragment_source = R"(
            #version 330 core

            in vec3 o_curv_pose;

            out vec4 FragColor;

            void main() {
                FragColor = vec4(o_curv_pose, 1.0f);
            }
        )";

        // methods

        StateEstimator_Impl::StateEstimator_Impl(std::mutex& mutex)
            : m_mutex {mutex}, m_curv_frame_lookup_mapped {false} {
            std::lock_guard<std::mutex> guard {mutex};

#ifdef DISPLAY
            m_gl_window = utils::create_sdl2_gl_window(
                "Spline Frame Lookup", curv_frame_lookup_tex_width, curv_frame_lookup_tex_width,
                0, &m_gl_context
            );
#else
            // dummy window to create opengl context for curv frame buffer
            m_gl_window = utils::create_sdl2_gl_window(
                "Spline Frame Lookup Dummy", 1, 1,
                SDL_WINDOW_HIDDEN, &m_gl_context
            );
#endif

            utils::make_gl_current_or_except(m_gl_window, m_gl_context);

            m_gl_path_shader = utils::compile_shader(vertex_source, fragment_source);

            glClearColor(0.0f, 0.0f, 0.0f, -1.0f);

            glEnable(GL_DEPTH_TEST);
            glDepthFunc(GL_LESS);

            glViewport(0, 0, curv_frame_lookup_tex_width, curv_frame_lookup_tex_width);

            gen_curv_frame_lookup_framebuffer();
            gen_gl_path();

            glFinish();
            utils::make_gl_current_or_except(m_gl_window, nullptr);
        }

        void StateEstimator_Impl::gen_curv_frame_lookup_framebuffer() {
            glGenFramebuffers(1, &m_curv_frame_lookup_fbo);
            glBindFramebuffer(GL_FRAMEBUFFER, m_curv_frame_lookup_fbo);

            glGenRenderbuffers(1, &m_curv_frame_lookup_rbo);
            glBindRenderbuffer(GL_RENDERBUFFER, m_curv_frame_lookup_rbo);
            glRenderbufferStorage(GL_RENDERBUFFER, GL_RGBA32F, curv_frame_lookup_tex_width, curv_frame_lookup_tex_width);
            glFramebufferRenderbuffer(GL_FRAMEBUFFER, GL_COLOR_ATTACHMENT0, GL_RENDERBUFFER, m_curv_frame_lookup_rbo);

            GLuint depth_rbo;
            glGenRenderbuffers(1, &depth_rbo);
            glBindRenderbuffer(GL_RENDERBUFFER, depth_rbo);
            glRenderbufferStorage(GL_RENDERBUFFER, GL_DEPTH_COMPONENT32, curv_frame_lookup_tex_width,  curv_frame_lookup_tex_width);
            glFramebufferRenderbuffer(GL_FRAMEBUFFER, GL_DEPTH_ATTACHMENT, GL_RENDERBUFFER, depth_rbo);

            if (glCheckFramebufferStatus(GL_FRAMEBUFFER) != GL_FRAMEBUFFER_COMPLETE) {
                throw std::runtime_error("Framebuffer is not complete");
            }
        }

        StateEstimator_Impl::~StateEstimator_Impl() {
            SDL_QuitSubSystem(SDL_INIT_VIDEO);
        }

        void StateEstimator_Impl::on_spline(const SplineMsg& spline_msg) {
            std::lock_guard<std::mutex> guard {m_mutex};

            std::cout << "------- ON SPLINE -----" << std::endl;

            utils::make_gl_current_or_except(m_gl_window, m_gl_context);

            m_spline_frames.clear();
            m_spline_frames.reserve(spline_msg.frames.size());

            for (const auto& frame : spline_msg.frames) {
                m_spline_frames.push_back({
                    static_cast<float>(frame.x),
                    static_cast<float>(frame.y)
                });
            }

            std::cout << "generating spline frame lookup texture info..." << std::endl;
            gen_tex_info({m_world_state[state_x_idx], m_world_state[state_y_idx]});
            std::cout << "xcenter: " << m_curv_frame_lookup_tex_info.xcenter
                      << " ycenter: " << m_curv_frame_lookup_tex_info.ycenter <<
                         " width: " << m_curv_frame_lookup_tex_info.width
            << std::endl;

            std::cout << "filling OpenGL buffers..." << std::endl;
            fill_path_buffers({m_world_state[state_x_idx], m_world_state[state_y_idx]});

            utils::sync_gl_and_unbind_context(m_gl_window);

            m_spline_ready = true;

            std::cout << "-------------------\n" << std::endl;
        }

        void StateEstimator_Impl::on_world_twist(const TwistMsg &twist_msg) {
            std::lock_guard<std::mutex> guard {m_mutex};

            const float yaw = m_world_state[state_yaw_idx];
            const float car_xdot = twist_msg.twist.linear.x * std::cos(yaw) + twist_msg.twist.linear.y * std::sin(yaw);
            const float car_ydot = -twist_msg.twist.linear.x * std::sin(yaw) + twist_msg.twist.linear.y * std::cos(yaw);
            const float car_yawdot = twist_msg.twist.angular.z;

            m_world_state[state_car_xdot_idx] = car_xdot;
            m_world_state[state_car_ydot_idx] = car_ydot;
            m_world_state[state_yawdot_idx] = car_yawdot;

            m_world_twist_ready = true;
        }

        void StateEstimator_Impl::on_world_quat(const QuatMsg &quat_msg) {
            using namespace glm;
            std::lock_guard<std::mutex> guard {m_mutex};

            const fquat quat = dquat(
                quat_msg.quaternion.w, quat_msg.quaternion.x, quat_msg.quaternion.y, quat_msg.quaternion.z
            );

            const fmat3x3 rot = mat3_cast(quat);
            const fvec3 ihatprime = rot * fvec3(1, 0, 0);
            const float yaw = std::atan2(ihatprime.y, ihatprime.x);

            m_world_state[state_yaw_idx] = yaw;

            m_world_yaw_ready = true;
        }

        // const float w = quat_msg.quaternion.w;
        // const float x = quat_msg.quaternion.x;
        // const float y = quat_msg.quaternion.y;
        // const float z = quat_msg.quaternion.z;

        // const float yaw = std::atan2(2 * (w * z + x * y), 1 - 2 * (y * y + z * z));

        void StateEstimator_Impl::on_world_pose(const PoseMsg &pose_msg) {
            std::lock_guard<std::mutex> guard {m_mutex};

            m_world_state[state_x_idx] = pose_msg.pose.position.x;
            m_world_state[state_y_idx] = pose_msg.pose.position.y;
            m_world_state[state_yaw_idx] = pose_msg.pose.orientation.z;

            m_world_yaw_ready = true;
        }

        void StateEstimator_Impl::on_state(const StateMsg& state_msg) {
            std::lock_guard<std::mutex> guard {m_mutex};

            std::cout << "------- ON STATE -----" << std::endl;

            m_world_state[state_x_idx] = state_msg.x;
            m_world_state[state_y_idx] = state_msg.y;
            m_world_state[state_yaw_idx] = state_msg.yaw;
            m_world_state[state_car_xdot_idx] = state_msg.xcar_dot;
            m_world_state[state_car_ydot_idx] = state_msg.ycar_dot;
            m_world_state[state_yawdot_idx] = state_msg.yaw_dot;
            m_world_state[state_my_idx] = state_msg.moment_y;
            m_world_state[state_fz_idx] = state_msg.downforce;
            m_world_state[state_whl_speed_f_idx] = state_msg.whl_speed_f;
            m_world_state[state_whl_speed_r_idx] = state_msg.whl_speed_r;

            m_world_twist_ready = true;
            m_world_yaw_ready = true;

            std::cout << "-------------------\n" << std::endl;
        }

        void StateEstimator_Impl::sync_to_device(float swangle) {
            std::lock_guard<std::mutex> guard {m_mutex};

            std::cout << "Publishing state" << std::endl;
            for (float dim : m_world_state)
            {
                std::cout << dim << " ";
            }

            // TODO: make wheel speed estimation optional
            estimate_whl_speeds(swangle);

            utils::make_gl_current_or_except(m_gl_window, m_gl_context);

            std::cout << "unmapping CUDA curv frame lookup texture for OpenGL rendering ..." << std::endl;
            unmap_curv_frame_lookup();

            std::cout << "rendering curv frame lookup table..." << std::endl;
            render_curv_frame_lookup();

            std::cout << "mapping OpenGL curv frame texture back to CUDA..." << std::endl;
            map_curv_frame_lookup();

            std::cout << "syncing world state to device..." << std::endl;
            sync_world_state();

            std::cout << "syncing spline frame lookup texture info to device..." << std::endl;
            sync_tex_info();

            utils::sync_gl_and_unbind_context(m_gl_window);
        }

        bool StateEstimator_Impl::is_ready() {
            return m_spline_ready && m_world_twist_ready && m_world_yaw_ready;
        }

#ifdef DISPLAY
        std::vector<glm::fvec2> StateEstimator_Impl::get_spline_frames() {
            std::lock_guard<std::mutex> guard {m_mutex};

            std::vector<glm::fvec2> res (m_spline_frames.size());
            for (size_t i = 0; i < m_spline_frames.size(); i++) {
                res[i] = {m_spline_frames[i].x, m_spline_frames[i].y};
            }
            return res;
        }

        void StateEstimator_Impl::get_offset_pixels(OffsetImage &offset_image) {
            std::lock_guard<std::mutex> guard {m_mutex};

            SDL_GLContext prev_context = SDL_GL_GetCurrentContext();
            SDL_Window* prev_window = SDL_GL_GetCurrentWindow();

            utils::make_gl_current_or_except(m_gl_window, m_gl_context);

            offset_image.pixels = std::vector<float>(curv_frame_lookup_tex_width * curv_frame_lookup_tex_width);
            glBindFramebuffer(GL_READ_FRAMEBUFFER, m_curv_frame_lookup_fbo);
            glReadPixels(
                0, 0, curv_frame_lookup_tex_width, curv_frame_lookup_tex_width,
                GL_GREEN, GL_FLOAT,
                offset_image.pixels.data()
            );

            offset_image.pix_width = curv_frame_lookup_tex_width;
            offset_image.pix_height = curv_frame_lookup_tex_width;
            offset_image.center = {m_curv_frame_lookup_tex_info.xcenter, m_curv_frame_lookup_tex_info.ycenter};
            offset_image.world_width = m_curv_frame_lookup_tex_info.width;

            utils::sync_gl_and_unbind_context(m_gl_window);
            utils::make_gl_current_or_except(prev_window, prev_context);
        }
#endif

        void StateEstimator_Impl::sync_world_state() {
            CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(
                cuda_globals::curr_state), &m_world_state, state_dims * sizeof(float)
            ));
        }

        void StateEstimator_Impl::sync_tex_info() {
            CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(
                cuda_globals::curv_frame_lookup_tex_info), &m_curv_frame_lookup_tex_info, sizeof(cuda_globals::CurvFrameLookupTexInfo)
            ));
        }

        void StateEstimator_Impl::gen_tex_info(glm::fvec2 car_pos) {
            float xmin = car_pos.x;
            float ymin = car_pos.y;
            float xmax = car_pos.x;
            float ymax = car_pos.y;

            for (const glm::fvec2 frame : m_spline_frames) {
                xmin = std::min(xmin, frame.x);
                xmax = std::max(xmax, frame.x);
                ymin = std::min(ymin, frame.y);
                ymax = std::max(ymax, frame.y);
            }

            m_curv_frame_lookup_tex_info.xcenter = (xmax + xmin) / 2;
            m_curv_frame_lookup_tex_info.ycenter = (ymax + ymin) / 2;
            m_curv_frame_lookup_tex_info.width = std::max(xmax - xmin, ymax - ymin) + curv_frame_lookup_padding * 2;
        }

        void StateEstimator_Impl::render_curv_frame_lookup() {
            glBindFramebuffer(GL_FRAMEBUFFER, m_curv_frame_lookup_fbo);

            glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
            glUseProgram(m_gl_path_shader);
            glUniform1f(shader_scale_loc, 2.0f / m_curv_frame_lookup_tex_info.width);
            glUniform2f(shader_center_loc, m_curv_frame_lookup_tex_info.xcenter, m_curv_frame_lookup_tex_info.ycenter);

            glBindVertexArray(m_gl_path.vao);
            glDrawElements(GL_TRIANGLES, (m_spline_frames.size() * 6 - 2) * 3, GL_UNSIGNED_INT, nullptr);

#ifdef DISPLAY
            glBindFramebuffer(GL_READ_FRAMEBUFFER, m_curv_frame_lookup_fbo);
            glBindFramebuffer(GL_DRAW_FRAMEBUFFER, 0);
            glBlitFramebuffer(
                0, 0, curv_frame_lookup_tex_width, curv_frame_lookup_tex_width,
                0, 0, curv_frame_lookup_tex_width, curv_frame_lookup_tex_width,
                GL_COLOR_BUFFER_BIT, GL_NEAREST
            );

            SDL_GL_SwapWindow(m_gl_window);
#endif
        }

        void StateEstimator_Impl::map_curv_frame_lookup() {
            CUDA_CALL(hipGraphicsGLRegisterImage(&m_curv_frame_lookup_rsc, m_curv_frame_lookup_rbo, GL_RENDERBUFFER, hipGraphicsRegisterFlagsNone));

            if (!m_curv_frame_lookup_mapped) {
                m_curv_frame_lookup_mapped = true;

                CUDA_CALL(hipGraphicsMapResources(1, &m_curv_frame_lookup_rsc));
            }

            hipResourceDesc img_rsc_desc {};
            img_rsc_desc.resType = hipResourceTypeMipmappedArray;
            CUDA_CALL(cudaGraphicsResourceGetMappedMipmappedArray(&img_rsc_desc.res.mipmap.mipmap, m_curv_frame_lookup_rsc));

            hipTextureDesc img_tex_desc {};
            img_tex_desc.addressMode[0] = hipAddressModeClamp;
            img_tex_desc.addressMode[1] = hipAddressModeClamp;
            img_tex_desc.filterMode = hipFilterModeLinear;
            img_tex_desc.readMode = hipReadModeElementType;
            img_tex_desc.normalizedCoords = true;

            hipTextureObject_t tex;
            CUDA_CALL(hipCreateTextureObject(&tex, &img_rsc_desc, &img_tex_desc, nullptr));
            CUDA_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(
                cuda_globals::curv_frame_lookup_tex), &tex, sizeof(hipTextureObject_t)
            ));
        }

        void StateEstimator_Impl::unmap_curv_frame_lookup() {
            if (!m_curv_frame_lookup_mapped)
                return;

            CUDA_CALL(hipGraphicsUnmapResources(1, &m_curv_frame_lookup_rsc));
            m_curv_frame_lookup_mapped = false;
        }

        void StateEstimator_Impl::gen_gl_path() {
            glGenVertexArrays(1, &m_gl_path.vao);
            glGenBuffers(1, &m_gl_path.vbo);
            glGenBuffers(1, &m_gl_path.ebo);

            glBindVertexArray(m_gl_path.vao);
            glBindBuffer(GL_ARRAY_BUFFER, m_gl_path.vbo);
            glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)0);
            glEnableVertexAttribArray(0);
            glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, 5 * sizeof(float), (void*)(2 * sizeof(float)));
            glEnableVertexAttribArray(1);
            glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, m_gl_path.ebo);

            glBindVertexArray(0);
        }

        void StateEstimator_Impl::fill_path_buffers(glm::fvec2 car_pos) {
            struct Vertex {
                struct {
                    float x;
                    float y;
                } world;

                struct {
                    float progress;
                    float offset;
                    float heading;
                } curv;
            };

            const float radius = track_width * 0.5f;
            const size_t n = m_spline_frames.size();

            std::vector<Vertex> vertices;
            std::vector<GLuint> indices;

            float total_progress = 0;
            for (size_t i = 0; i < n - 1; i++) {
                glm::fvec2 p1 = m_spline_frames[i];
                glm::fvec2 p2 = m_spline_frames[i + 1];

                glm::fvec2 disp = p2 - p1;
                float new_progress = glm::length(disp);
                float segment_heading = std::atan2(disp.y, disp.x);


                glm::fvec2 prev = i == 0 ? p1 : m_spline_frames[i - 1];
                float secant_heading = std::atan2(p2.y - prev.y, p2.x - prev.x);

                glm::fvec2 dir = glm::normalize(disp);
                glm::fvec2 normal = glm::fvec2(-dir.y, dir.x);

                glm::fvec2 low1 = p1 - normal * radius;
                glm::fvec2 low2 = p2 - normal * radius;
                glm::fvec2 high1 = p1 + normal * radius;
                glm::fvec2 high2 = p2 + normal * radius;

                if (i == 0) {
                    vertices.push_back({{p1.x, p1.y}, {total_progress, 0.0f, segment_heading}});
                }
                vertices.push_back({{p2.x, p2.y}, {total_progress + new_progress, 0.0f, secant_heading}});

                vertices.push_back({{low1.x, low1.y}, {total_progress, -radius, segment_heading}});
                vertices.push_back({{low2.x, low2.y}, {total_progress + new_progress, -radius, segment_heading}});
                vertices.push_back({{high1.x, high1.y}, {total_progress, radius, segment_heading}});
                vertices.push_back({{high2.x, high2.y}, {total_progress + new_progress, radius, segment_heading}});

                const GLuint p1i = i == 0 ? 0 : (i - 1) * 5 + 1;
                const GLuint p2i = i * 5 + 1;
                const GLuint l1i = i * 5 + 2;
                const GLuint l2i = i * 5 + 3;
                const GLuint h1i = i * 5 + 4;
                const GLuint h2i = i * 5 + 5;

                indices.push_back(p1i);
                indices.push_back(p2i);
                indices.push_back(h2i);

                indices.push_back(h1i);
                indices.push_back(p1i);
                indices.push_back(h2i);

                indices.push_back(l1i);
                indices.push_back(l2i);
                indices.push_back(p2i);

                indices.push_back(p1i);
                indices.push_back(l1i);
                indices.push_back(p2i);

                if (i > 0) {
                    const GLuint lpi = (i - 1) * 5 + 3;
                    const GLuint hpi = (i - 1) * 5 + 5;

                    indices.push_back(hpi);
                    indices.push_back(p1i);
                    indices.push_back(h1i);

                    indices.push_back(lpi);
                    indices.push_back(l1i);
                    indices.push_back(p1i);
                }

                total_progress += new_progress;
            }

            // allow car to be before first frame
            {
                const GLuint ai = 2;
                const GLuint bi = 0;
                const GLuint ci = 4;

                const glm::fvec2 a = {vertices[ai].world.x, vertices[ai].world.y};
                const glm::fvec2 b = {vertices[bi].world.x, vertices[bi].world.y};
                const glm::fvec2 c = {vertices[ci].world.x, vertices[ci].world.y};

                const glm::fvec2 ac_unit = glm::normalize(c - a);
                const glm::fvec2 ac_norm = glm::fvec2(ac_unit.y, -ac_unit.x);

                if (glm::dot(car_pos - b, ac_norm) < 0) { // car is behind first triangles
                    const glm::fvec2 bcar = car_pos - b;
                    const glm::fvec2 car_parallel_plane = glm::normalize(glm::fvec2(bcar.y, -bcar.x));
                    const glm::fvec2 new_edge_center = b + bcar * (glm::length(bcar) + car_padding) / glm::length(bcar);

                    const glm::fvec2 v1_world = new_edge_center - car_parallel_plane * radius;
                    const glm::fvec2 v2_world = new_edge_center + car_parallel_plane * radius;

                    const float v1_progress = glm::dot( v1_world - b, ac_norm);
                    const float v2_progress = glm::dot(v2_world - b, ac_norm);

                    const float v1_offset = glm::dot(v1_world - b, ac_unit);
                    const float v2_offset = glm::dot(v2_world - b, ac_unit);

                    const float v1_heading = vertices[bi].curv.heading;
                    const float v2_heading = vertices[bi].curv.heading;

                    const Vertex v1 = {{v1_world.x, v1_world.y}, {v1_progress, v1_offset, v1_heading}};
                    const Vertex v2 = {{v2_world.x, v2_world.y}, {v2_progress, v2_offset, v2_heading}};

                    vertices.push_back(v1);
                    vertices.push_back(v2);

                    const GLuint v1i = vertices.size() - 2;
                    const GLuint v2i = vertices.size() - 1;

                    indices.push_back(v1i);
                    indices.push_back(ai);
                    indices.push_back(ci);

                    indices.push_back(v1i);
                    indices.push_back(ci);
                    indices.push_back(v2i);
                }
            }


            glBindBuffer(GL_ARRAY_BUFFER, m_gl_path.vbo);
            glBufferData(GL_ARRAY_BUFFER, sizeof(Vertex) * vertices.size(), vertices.data(), GL_DYNAMIC_DRAW);

            glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, m_gl_path.ebo);
            glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(GLuint) * indices.size(), indices.data(), GL_DYNAMIC_DRAW);
        }

        void StateEstimator_Impl::estimate_whl_speeds(float swangle) {
            const float xdot = m_world_state[state_car_xdot_idx];
            const float yawdot = m_world_state[state_yawdot_idx];

            const float whl_speed_f = (xdot * std::cos(swangle) + cg_to_front * yawdot * std::sin(swangle)) / whl_radius;
            const float whl_speed_r = xdot / whl_radius;

            m_world_state[state_whl_speed_f_idx] = whl_speed_f;
            m_world_state[state_whl_speed_r_idx] = whl_speed_r;
            std::cout << "whl_speed_f: " << whl_speed_f << " whl_speed_r: " << whl_speed_r << std::endl;
        }
    }
}
