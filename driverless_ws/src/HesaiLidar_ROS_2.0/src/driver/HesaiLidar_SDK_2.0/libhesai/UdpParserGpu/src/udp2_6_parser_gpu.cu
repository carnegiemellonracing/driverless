#include "hip/hip_runtime.h"
/************************************************************************************************
Copyright (C) 2023 Hesai Technology Co., Ltd.
Copyright (C) 2023 Original Authors
All rights reserved.

All code in this repository is released under the terms of the following Modified BSD License. 
Redistribution and use in source and binary forms, with or without modification, are permitted 
provided that the following conditions are met:

* Redistributions of source code must retain the above copyright notice, this list of conditions and 
  the following disclaimer.

* Redistributions in binary form must reproduce the above copyright notice, this list of conditions and 
  the following disclaimer in the documentation and/or other materials provided with the distribution.

* Neither the name of the copyright holder nor the names of its contributors may be used to endorse or 
  promote products derived from this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR IMPLIED 
WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A 
PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR 
ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT 
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR 
TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF 
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
************************************************************************************************/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_device_runtime_api.h>

#include "udp2_6_parser_gpu.h"
#include "safe_call.cuh"
#include "return_code.h"

using namespace hesai::lidar;
template <typename T_Point>
Udp2_6ParserGpu<T_Point>::Udp2_6ParserGpu() {
  corrections_loaded_ = false;
  cudaSafeMalloc(point_data_cu_, POINT_DATA_LEN);
  cudaSafeMalloc(sensor_timestamp_cu_, SENSOR_TIMESTAMP_LEN);
}
template <typename T_Point>
Udp2_6ParserGpu<T_Point>::~Udp2_6ParserGpu() {
  cudaSafeFree(point_data_cu_);
  cudaSafeFree(sensor_timestamp_cu_);
  if (corrections_loaded_) {
    cudaSafeFree(channel_elevations_cu_);
    cudaSafeFree(channel_azimuths_cu_);
    cudaSafeFree(channel_azimuths_adjust_cu_);
    cudaSafeFree(channel_elevations_adjust_cu_);
    corrections_loaded_ = false;
  }
}
template <typename T_Point>
__global__ void compute_xyzs_2_6_impl(T_Point *xyzs, const float* channel_azimuths, const float* channel_elevations,
    const float* azimuth_adjust, const float* elevation_adjust, const uint8_t azimuth_adjust_interval, const uint8_t elevation_adjust_interval,
    const PointDecodeData* point_data, const uint64_t* sensor_timestamp, const double raw_distance_unit, Transform transform, const uint32_t per_points_num, const uint16_t packet_index) {
  auto iscan = blockIdx.x;
  auto ichannel = threadIdx.x;
  if (iscan >= packet_index || ichannel >= per_points_num) return;
  int point_index = iscan * per_points_num + (ichannel % per_points_num);
  float apha =  channel_elevations[0];
  float beta =  channel_elevations[1];
  float gamma =  channel_elevations[2];
  float raw_azimuth = point_data[point_index].azimuth;
  float raw_elevation = point_data[point_index].elevation;
  float phi = channel_azimuths[point_data[point_index].chn_index + 3];
  float theta = channel_elevations[point_data[point_index].chn_index + 3];
  float an = apha + phi;
  float theta_n = (raw_elevation + theta / std::cos(an * M_PI / 180));
  float elv_v = raw_elevation * M_PI / 180 + theta * M_PI / 180 - std::tan(raw_elevation * M_PI / 180) * (1 - std::cos(an * M_PI / 180)) ;
  float delt_azi_v = std::sin(an * M_PI / 180) * std::cos(an * M_PI / 180) * theta_n * theta_n / 2  * 1.016 * M_PI / 180 * M_PI / 180;
  float eta = phi + delt_azi_v * 180 / M_PI + beta + raw_azimuth / 2;
  float delt_azi_h = std::sin(eta * M_PI / 180) * std::tan(2 * gamma * M_PI / 180) * std::tan(elv_v ) + std::sin(2 * eta * M_PI / 180) * gamma * gamma * M_PI / 180 * M_PI / 180;
  float elv_h = elv_v * 180 / M_PI + std::cos(eta * M_PI / 180) * 2 * gamma ;
  float azi_h = 90 +  raw_azimuth + delt_azi_h * 180 / M_PI + delt_azi_v * 180 / M_PI + phi;

  if (azimuth_adjust_interval != 0 && elevation_adjust_interval != 0) {
    float azi_a = azi_h - 90;
    float azimuth_fov = 120.0f;
    float elevation_fov = 25.0f;
    float adjust_interval_resolution = 0.5f;
    int azimuth_offset_num = int(azimuth_fov / (azimuth_adjust_interval * adjust_interval_resolution) + 1);
    int elevation_offset_num = int(elevation_fov / (elevation_adjust_interval * adjust_interval_resolution) + 1);
    int offset_index1 = int((azi_a + azimuth_fov / 2) / (azimuth_adjust_interval * adjust_interval_resolution));      //azi dimension
    int offset_index2 = int((elv_h + elevation_fov / 2) / (elevation_adjust_interval * adjust_interval_resolution));      //ele dimension
    if (offset_index1 >= 0 && offset_index1 < (azimuth_offset_num - 1) && offset_index2 >= 0 && offset_index2 < (elevation_offset_num - 1)) {
      float coefficient1 = ((offset_index1 + 1) * (azimuth_adjust_interval * adjust_interval_resolution) - azi_a - azimuth_fov / 2) / (azimuth_adjust_interval * adjust_interval_resolution);
      float coefficient2 = ((offset_index2 + 1) * (elevation_adjust_interval * adjust_interval_resolution) - elv_h - elevation_fov / 2) / (elevation_adjust_interval * adjust_interval_resolution);
      float offset1 = coefficient1 * azimuth_adjust[offset_index1  + offset_index2 * azimuth_offset_num] + (1 - coefficient1) * azimuth_adjust[offset_index1 + 1 + offset_index2 * azimuth_offset_num];
      float offset2 = coefficient1 * azimuth_adjust[offset_index1 + (offset_index2 + 1) * azimuth_offset_num] + (1 - coefficient1) * azimuth_adjust[offset_index1 + 1 + (offset_index2 + 1) * azimuth_offset_num];
      azi_h += (coefficient2 * offset1 + (1 - coefficient2) * offset2);
    }
    azi_a = azi_h - 90;
    offset_index1 = int((azi_a + azimuth_fov / 2) / (azimuth_adjust_interval * adjust_interval_resolution));
    if (offset_index1 >= 0 && offset_index1 < (azimuth_offset_num - 1) && offset_index2 >= 0 && offset_index2 < (elevation_offset_num - 1)) {
      float coefficient1 = ((offset_index1 + 1) * (azimuth_adjust_interval * adjust_interval_resolution) - azi_a - azimuth_fov / 2) / (azimuth_adjust_interval * adjust_interval_resolution);
      float coefficient2 = ((offset_index2 + 1) * (elevation_adjust_interval * adjust_interval_resolution) - elv_h - elevation_fov / 2) / (elevation_adjust_interval * adjust_interval_resolution);
      float offset1 = coefficient1 * elevation_adjust[offset_index1  + offset_index2 * azimuth_offset_num] + (1 - coefficient1) * elevation_adjust[offset_index1 + 1 + offset_index2 * azimuth_offset_num];
      float offset2 = coefficient1 * elevation_adjust[offset_index1 + (offset_index2 + 1) * azimuth_offset_num] + (1 - coefficient1) * elevation_adjust[offset_index1 + 1 + (offset_index2 + 1) * azimuth_offset_num];
      elv_h += (coefficient2 * offset1 + (1 - coefficient2) * offset2);
    }
  }

  auto rho = point_data[point_index].distances * raw_distance_unit;
  float z = rho * std::sin(elv_h * M_PI / 180);
  auto r = rho * std::cos(elv_h * M_PI / 180) ;
  float x = r * std::sin(azi_h * M_PI / 180);
  float y = r * std::cos(azi_h * M_PI / 180);

  float cosa = std::cos(transform.roll);
  float sina = std::sin(transform.roll);
  float cosb = std::cos(transform.pitch);
  float sinb = std::sin(transform.pitch);
  float cosc = std::cos(transform.yaw);
  float sinc = std::sin(transform.yaw);

  float x_ = cosb * cosc * x + (sina * sinb * cosc - cosa * sinc) * y +
              (sina * sinc + cosa * sinb * cosc) * z + transform.x;
  float y_ = cosb * sinc * x + (cosa * cosc + sina * sinb * sinc) * y +
              (cosa * sinb * sinc - sina * cosc) * z + transform.y;
  float z_ = -sinb * x + sina * cosb * y + cosa * cosb * z + transform.z;
  gpu::setX(xyzs[point_index], x_);
  gpu::setY(xyzs[point_index],  y_);
  gpu::setZ(xyzs[point_index], z_);
  gpu::setIntensity(xyzs[point_index], point_data[point_index].reflectivities);
  gpu::setTimestamp(xyzs[point_index], double(sensor_timestamp[iscan]) / kMicrosecondToSecond);
  gpu::setRing(xyzs[point_index], point_data[point_index].chn_index);
}
template <typename T_Point>
int Udp2_6ParserGpu<T_Point>::ComputeXYZI(LidarDecodedFrame<T_Point> &frame) {
  if (!corrections_loaded_) return int(ReturnCode::CorrectionsUnloaded);          
  cudaSafeCall(hipMemcpy(point_data_cu_, frame.pointData,
                          frame.per_points_num * frame.packet_num * sizeof(PointDecodeData), 
                          hipMemcpyHostToDevice), ReturnCode::CudaMemcpyHostToDeviceError);
  cudaSafeCall(hipMemcpy(sensor_timestamp_cu_, frame.sensor_timestamp,
                          frame.packet_num * sizeof(uint64_t), 
                          hipMemcpyHostToDevice), ReturnCode::CudaMemcpyHostToDeviceError);                             
compute_xyzs_2_6_impl<<<frame.packet_num, frame.per_points_num>>>(this->frame_.gpu()->points, channel_azimuths_cu_, channel_elevations_cu_, 
  channel_azimuths_adjust_cu_, channel_elevations_adjust_cu_, corrections_.azimuth_adjust_interval, corrections_.elevation_adjust_interval,
  point_data_cu_, sensor_timestamp_cu_, frame.distance_unit, this->transform_, frame.per_points_num, frame.packet_num);
  cudaSafeCall(hipGetLastError(), ReturnCode::CudaXYZComputingError);
  this->frame_.DeviceToHost(0, frame.per_points_num * frame.packet_num * sizeof(T_Point));
  std::memcpy(frame.points, this->frame_.cpu()->points, frame.per_points_num * frame.packet_num * sizeof(T_Point));
  return 0;
}
template<typename T_Point>
int Udp2_6ParserGpu<T_Point>::LoadCorrectionString(char *data) {
  if (LoadCorrectionDatData(data) == 0) {
    return 0;
  }
  return LoadCorrectionCsvData(data);
}


// csv ----> correction
template<typename T_Point>
int  Udp2_6ParserGpu<T_Point>::LoadCorrectionCsvData(char *correction_string)
{
  std::string correction_content_str = correction_string;
  std::istringstream ifs(correction_content_str);
  std::string line;

  // skip first line "Laser id,Elevation,Azimuth" or "eeff"
  std::getline(ifs, line);  
  float elevation_list[MAX_LASER_NUM], azimuth_list[MAX_LASER_NUM];
  std::vector<std::string> vfirstLine;
  split_string(vfirstLine, line, ',');
  if (vfirstLine[0] == "EEFF" || vfirstLine[0] == "eeff") {
    // skip second line
    std::getline(ifs, line);  
  }

  int lineCount = 0;
  while (std::getline(ifs, line)) {
    std::vector<std::string> vLineSplit;
    split_string(vLineSplit, line, ',');
    // skip error line or hash value line
    if (vLineSplit.size() < 3) {  
      continue;
    } else {
      lineCount++;
    }
    float elevation, azimuth;
    int laserId = 0;

    std::stringstream ss(line);
    std::string subline;
    std::getline(ss, subline, ',');
    std::stringstream(subline) >> laserId;
    std::getline(ss, subline, ',');
    std::stringstream(subline) >> elevation;
    std::getline(ss, subline, ',');
    std::stringstream(subline) >> azimuth;

    if (laserId > MAX_LASER_NUM || laserId <= 0) {
      LogFatal("laser id is wrong in correction file. laser Id: %d, line: %d", laserId, lineCount);
      continue;
    }
    if (laserId != lineCount) {
      LogWarning("laser id is wrong in correction file. laser Id: %d, line: %d.  continue", laserId, lineCount);
      continue;
    }
    elevation_list[laserId - 1] = elevation;
    azimuth_list[laserId - 1] = azimuth;
  }

  for (int i = 0; i < lineCount; ++i) {
    corrections_.azimuths[i] = azimuth_list[i];
    corrections_.elevations[i] = elevation_list[i];
  }
  CUDACheck(hipMalloc(&channel_azimuths_cu_, sizeof(corrections_.azimuths)));
  CUDACheck(hipMalloc(&channel_elevations_cu_, sizeof(corrections_.azimuths)));
  CUDACheck(hipMemcpy(channel_azimuths_cu_, corrections_.azimuths, sizeof(corrections_.azimuths), hipMemcpyHostToDevice));
  CUDACheck(hipMemcpy(channel_elevations_cu_, corrections_.elevations, sizeof(corrections_.azimuths), hipMemcpyHostToDevice));
  corrections_loaded_ = true;
  return 0;
}


// buffer(.bin) ---> correction
template<typename T_Point>
int Udp2_6ParserGpu<T_Point>::LoadCorrectionDatData(char *data) {
  try {
    char *p = data;
    struct ETCorrectionsHeader ETheader = *((struct ETCorrectionsHeader* )p);
    if (0xee == ETheader.delimiter[0] && 0xff == ETheader.delimiter[1]) {
      switch (ETheader.min_version) {
        case 1: {
          ETCorrectionsHeader_V1V2 correction_v1;
          memcpy((void *)&correction_v1, p, sizeof(struct ETCorrectionsHeader_V1V2));
          corrections_.header.getDataFromV1V2(correction_v1);
          p += sizeof(ETCorrectionsHeader_V1V2);
          auto channel_num = corrections_.header.channel_number;
          uint16_t division = corrections_.header.angle_division;
          if ((channel_num > ET_MAX_CHANNEL_NUM - 3) || division == 0) {
            LogError("data error: channel_num is %u, division is %u", channel_num, division);
            return -1;
          }
          memcpy((void *)&corrections_.raw_azimuths, p,
                 sizeof(int16_t) * channel_num);
          p += sizeof(int16_t) * channel_num;
          memcpy((void *)&corrections_.raw_elevations, p,
                 sizeof(int16_t) * channel_num);
          p += sizeof(uint32_t) * channel_num;
          corrections_.elevations[0] = ((float)(corrections_.header.apha)) / division;
          corrections_.elevations[1] = ((float)(corrections_.header.beta)) / division;
          corrections_.elevations[2] = ((float)(corrections_.header.gamma)) / division;
          LogDebug("apha:%f, beta:%f, gamma:%f", corrections_.elevations[0], corrections_.elevations[1], corrections_.elevations[2]);
          for (int i = 0; i < channel_num; i++) {
            corrections_.azimuths[i + 3] = ((float)(corrections_.raw_azimuths[i])) / division;
            corrections_.elevations[i + 3] = ((float)(corrections_.raw_elevations[i])) / division;
            LogDebug("%d %f %f",i, corrections_.azimuths[i + 3], corrections_.elevations[i + 3]);
          }
          corrections_.azimuth_adjust_interval = 0;
          corrections_.elevation_adjust_interval = 0;
          
          memcpy((void*)&corrections_.SHA_value, p, 32);
          // successed
          CUDACheck(hipMalloc(&channel_azimuths_cu_, sizeof(corrections_.azimuths)));
          CUDACheck(hipMalloc(&channel_elevations_cu_, sizeof(corrections_.elevations)));
          CUDACheck(hipMalloc(&channel_azimuths_adjust_cu_, sizeof(corrections_.azimuth_adjust_f)));
          CUDACheck(hipMalloc(&channel_elevations_adjust_cu_, sizeof(corrections_.elevation_adjust_f)));
          CUDACheck(hipMemcpy(channel_azimuths_cu_, corrections_.azimuths, sizeof(corrections_.azimuths), hipMemcpyHostToDevice));
          CUDACheck(hipMemcpy(channel_elevations_cu_, corrections_.elevations, sizeof(corrections_.elevations), hipMemcpyHostToDevice));
          corrections_loaded_ = true;
          return 0;
        } break;
        case 2: {
          ETCorrectionsHeader_V1V2 correction_v1;
          memcpy((void *)&correction_v1, p, sizeof(struct ETCorrectionsHeader_V1V2));
          corrections_.header.getDataFromV1V2(correction_v1);
          p += sizeof(ETCorrectionsHeader_V1V2);
          auto channel_num = corrections_.header.channel_number;
          uint16_t division = corrections_.header.angle_division;
          if ((channel_num > ET_MAX_CHANNEL_NUM - 3) || division == 0) {
            LogError("data error: channel_num is %u, division is %u", channel_num, division);
            return -1;
          }
          memcpy((void *)&corrections_.raw_azimuths, p,
                 sizeof(int16_t) * channel_num);
          p += sizeof(int16_t) * channel_num;
          memcpy((void *)&corrections_.raw_elevations, p,
                 sizeof(int16_t) * channel_num);
          p += sizeof(int16_t) * channel_num;
          corrections_.elevations[0] = ((float)(corrections_.header.apha)) / division;
          corrections_.elevations[1] = ((float)(corrections_.header.beta)) / division;
          corrections_.elevations[2] = ((float)(corrections_.header.gamma)) / division;
          LogDebug("apha:%f, beta:%f, gamma:%f", corrections_.elevations[0], corrections_.elevations[1], corrections_.elevations[2]);
          for (int i = 0; i < channel_num; i++) {
            corrections_.azimuths[i + 3] = ((float)(corrections_.raw_azimuths[i])) / division;
            corrections_.elevations[i + 3] = ((float)(corrections_.raw_elevations[i])) / division;
            LogDebug("%d %f %f",i, corrections_.azimuths[i + 3], corrections_.elevations[i + 3]);
          }
          corrections_.azimuth_adjust_interval = *((char*)p);
          p = p + 1;
          corrections_.elevation_adjust_interval = *((char*)p);
          p = p + 1;
          int angle_offset_len = (120 / (corrections_.azimuth_adjust_interval * 0.5) + 1) * (25 / (corrections_.elevation_adjust_interval * 0.5) + 1);
          memcpy((void*)corrections_.azimuth_adjust, p, sizeof(int16_t) * angle_offset_len);
          p = p + sizeof(int16_t) * angle_offset_len;
          memcpy((void*)corrections_.elevation_adjust, p, sizeof(int16_t) * angle_offset_len); 
          p = p + sizeof(int16_t) * angle_offset_len;
          for (int i = 0; i < angle_offset_len; i++) {
            corrections_.azimuth_adjust_f[i] = 1.f * corrections_.azimuth_adjust[i] / corrections_.header.angle_division;
            corrections_.elevation_adjust_f[i] = 1.f * corrections_.elevation_adjust[i] / corrections_.header.angle_division;
          }
          // int adjustNum = channel_num;
          memcpy((void*)&corrections_.SHA_value, p, 32);
          // successed
          CUDACheck(hipMalloc(&channel_azimuths_cu_, sizeof(corrections_.azimuths)));
          CUDACheck(hipMalloc(&channel_elevations_cu_, sizeof(corrections_.elevations)));
          CUDACheck(hipMalloc(&channel_azimuths_adjust_cu_, sizeof(corrections_.azimuth_adjust_f)));
          CUDACheck(hipMalloc(&channel_elevations_adjust_cu_, sizeof(corrections_.elevation_adjust_f)));
          CUDACheck(hipMemcpy(channel_azimuths_adjust_cu_, corrections_.azimuth_adjust_f, sizeof(corrections_.azimuth_adjust_f), hipMemcpyHostToDevice));
          CUDACheck(hipMemcpy(channel_elevations_adjust_cu_, corrections_.elevation_adjust_f, sizeof(corrections_.elevation_adjust_f), hipMemcpyHostToDevice));
          CUDACheck(hipMemcpy(channel_azimuths_cu_, corrections_.azimuths, sizeof(corrections_.azimuths), hipMemcpyHostToDevice));
          CUDACheck(hipMemcpy(channel_elevations_cu_, corrections_.elevations, sizeof(corrections_.elevations), hipMemcpyHostToDevice));
          corrections_loaded_ = true;
          return 0;
        } break;
        default:
          LogWarning("min_version is wrong!");
          break;
      }
    } else {
        return -1;
    }
  } catch (const std::exception &e) {
    LogFatal("load correction error: %s", e.what());
    return -1;
  }
  return -1;
}
template <typename T_Point>
int Udp2_6ParserGpu<T_Point>::LoadCorrectionFile(std::string lidar_correction_file) {
  LogInfo("load correction file from local correction.csv now!");
  std::ifstream fin(lidar_correction_file);
  if (fin.is_open()) {
    LogDebug("Open correction file success");
    int length = 0;
    fin.seekg(0, std::ios::end);
    length = fin.tellg();
    fin.seekg(0, std::ios::beg);
    char *buffer = new char[length];
    fin.read(buffer, length);
    fin.close();
    int ret = LoadCorrectionString(buffer);
    delete[] buffer;
    if (ret != 0) {
      LogError("Parse local Correction file Error");
    } else {
      LogInfo("Parse local Correction file Success!!!");
      return 0;
    }
  } else {
    LogError("Open correction file failed");
    return -1;
  }
  return -1;
}

